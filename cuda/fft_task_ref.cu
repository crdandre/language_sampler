//====================================================
// CUDA FFT Task Reference
//====================================================

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hiprand/hiprand.h>
#include <stdio.h>
#include <math.h>

// Constants
#define BLOCK_SIZE 256
#define SIGNAL_LENGTH 1024
#define WINDOW_SIZE 8
#define PI 3.14159265359f

// Error checking macro
#define cudaCheckError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

// Kernel for generating sine wave
__global__ void generateSignalKernel(float* signal, float frequency, float sampleRate) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < SIGNAL_LENGTH) {
        float t = idx / sampleRate;
        signal[idx] = sinf(2.0f * PI * frequency * t);
    }
}

// Kernel for moving average filter
__global__ void movingAverageKernel(float* input, float* output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < SIGNAL_LENGTH - WINDOW_SIZE + 1) {
        float sum = 0.0f;
        for (int i = 0; i < WINDOW_SIZE; i++) {
            sum += input[idx + i];
        }
        output[idx] = sum / WINDOW_SIZE;
    }
}

// Kernel for peak finding
__global__ void findPeaksKernel(float* signal, int* peaks, float threshold) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx > 0 && idx < SIGNAL_LENGTH - 1) {
        if (signal[idx] > threshold &&
            signal[idx] > signal[idx-1] &&
            signal[idx] > signal[idx+1]) {
            peaks[idx] = 1;
        } else {
            peaks[idx] = 0;
        }
    }
}

int main() {
    // Host arrays
    float *h_signal = nullptr;
    float *h_filtered = nullptr;
    float *h_spectrum = nullptr;
    int *h_peaks = nullptr;

    // Device arrays
    float *d_signal = nullptr;
    float *d_filtered = nullptr;
    float *d_spectrum = nullptr;
    int *d_peaks = nullptr;

    // Allocate host memory
    h_signal = (float*)malloc(SIGNAL_LENGTH * sizeof(float));
    h_filtered = (float*)malloc(SIGNAL_LENGTH * sizeof(float));
    h_spectrum = (float*)malloc(SIGNAL_LENGTH * sizeof(float));
    h_peaks = (int*)malloc(SIGNAL_LENGTH * sizeof(int));

    // Allocate device memory
    cudaCheckError(hipMalloc(&d_signal, SIGNAL_LENGTH * sizeof(float)));
    cudaCheckError(hipMalloc(&d_filtered, SIGNAL_LENGTH * sizeof(float)));
    cudaCheckError(hipMalloc(&d_spectrum, SIGNAL_LENGTH * sizeof(float)));
    cudaCheckError(hipMalloc(&d_peaks, SIGNAL_LENGTH * sizeof(int)));

    // Calculate grid and block dimensions
    dim3 blockDim(BLOCK_SIZE);
    dim3 gridDim((SIGNAL_LENGTH + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Generate signal
    generateSignalKernel<<<gridDim, blockDim>>>(d_signal, 10.0f, 1000.0f);
    cudaCheckError(hipGetLastError());

    // Apply moving average filter
    movingAverageKernel<<<gridDim, blockDim>>>(d_signal, d_filtered);
    cudaCheckError(hipGetLastError());

    // Setup cuFFT
    hipfftHandle plan;
    hipfftPlan1d(&plan, SIGNAL_LENGTH, HIPFFT_R2C, 1);
    hipfftExecR2C(plan, (hipfftReal*)d_filtered, (hipfftComplex*)d_spectrum);
    
    // Find peaks
    findPeaksKernel<<<gridDim, blockDim>>>(d_spectrum, d_peaks, 0.5f);
    cudaCheckError(hipGetLastError());

    // Copy results back to host
    cudaCheckError(hipMemcpy(h_signal, d_signal, SIGNAL_LENGTH * sizeof(float), hipMemcpyDeviceToHost));
    cudaCheckError(hipMemcpy(h_filtered, d_filtered, SIGNAL_LENGTH * sizeof(float), hipMemcpyDeviceToHost));
    cudaCheckError(hipMemcpy(h_spectrum, d_spectrum, SIGNAL_LENGTH * sizeof(float), hipMemcpyDeviceToHost));
    cudaCheckError(hipMemcpy(h_peaks, d_peaks, SIGNAL_LENGTH * sizeof(int), hipMemcpyDeviceToHost));

    // Cleanup
    hipfftDestroy(plan);
    hipFree(d_signal);
    hipFree(d_filtered);
    hipFree(d_spectrum);
    hipFree(d_peaks);
    free(h_signal);
    free(h_filtered);
    free(h_spectrum);
    free(h_peaks);

    return 0;
}
